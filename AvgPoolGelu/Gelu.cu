#include "hip/hip_runtime.h"
#include "Gelu.cuh"
#include <stdio.h>
#include <iostream>

template <typename T> 
__global__ void geluKernel(T* c, const T* a, const int* b)
{
    T par2 = (T)0.5f;
    T par1 = (T)0.044715f;
    T sqrt2Divpi = (T)0.7978845608028654f;
    T one = (T)1.0f;

    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int matrixSize_i = b[0];
    int matrixSize_j = b[1];
    T sum = a[i + matrixSize_i * j];
    T x0 = sqrt2Divpi * (sum + par1 * (sum * sum * sum));
    T tanVal;
    tanVal = (T)tanhf(x0);
    c[i + matrixSize_i * j] = par2 * sum * (one + tanVal);
}


template <typename T>
hipError_t geluWithCuda<T>(T* c, T* a, int* b, unsigned int output_size_i, unsigned int output_size_j)
{
    T* dev_a = 0;
    int* dev_b = 0;
    T* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, output_size_i * output_size_j * sizeof(T));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, output_size_i * output_size_j * sizeof(T));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, 6 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, output_size_i * output_size_j * sizeof(T), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, 6 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    // avgGeluKernel<<<dim3(1,1), dim3(output_size_i, output_size_j)>>>(dev_c, dev_a, dev_b);

    geluKernel<T> <<<dim3(output_size_i, output_size_j), dim3(1, 1) >>> (dev_c, dev_a, dev_b);


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "avgPoolKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching avgGeluKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, output_size_i * output_size_j * sizeof(T), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

template hipError_t geluWithCuda<float>(float* c, float* a, int* b, unsigned int output_size_i, unsigned int output_size_j);
template hipError_t geluWithCuda<double>(double* c, double* a, int* b, unsigned int output_size_i, unsigned int output_size_j);
template hipError_t geluWithCuda<half>(half* c, half* a, int* b, unsigned int output_size_i, unsigned int output_size_j);

